#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#include "cudaAcceleration.h"
#ifdef USE_CUDA

#define PINNED
// includes, project
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "malloc_a.h"
#include "cudaAcceleration.h"
#include "cudaAcc_data.h"
//#ifndef __linux__ //Pretty sure this is unused on windows as well... Can we delete?
//#include "cudaAcc_CleanExit.h" // TODO: DELETE THAT, only for testing
//#endif // not __linux__

#include "cudaAcc_utilities.h"
#include "nvapi_device.h"

unsigned cmem_rtotal = 0;

float2* dev_cx_DataArray;
float2* dev_cx_ChirpDataArray;
//float2* float_CurrentTrig;
double2* dev_CurrentTrig;

float2* dev_WorkData;
float *dev_PowerSpectrum;
float *dev_t_PowerSpectrum;
float *dev_PoT;
float *dev_PoTPrefixSum;

hipStream_t fftstream0 = NULL;
hipStream_t fftstream1 = NULL;

#define CUDA_MAXNUMSTREAMS 2
int cudaAcc_NumDataPoints;
//int cudaps_blksize=64;
//int cuda_tmax=256;
bool cuda_pinned = false;
hipStream_t cudapsStream[CUDA_MAXNUMSTREAMS];
//hipStream_t cudaAutocorrStream[8];
hipStream_t cudaAutocorrStream;

extern __global__ void cudaAcc_summax32_kernel(float *input, float3* output, int iterations);
template <int blockx> __global__ void find_triplets_kernel(int ul_FftLength, int len_power, volatile float triplet_thresh, int AdvanceBy);
template <bool load_state, int num_adds> __global__ void find_pulse_kernel(float best_pulse_score, int PulsePotLen, int AdvanceBy, int fft_len, int ndivs);

float4* dev_GaussFitResults;
float4* dev_GaussFitResultsReordered;
float4* dev_GaussFitResultsReordered2;
float *dev_NormMaxPower;
result_flag* dev_flag;
float4* GaussFitResults;
float4* GaussFitResults2;

float *dev_flagged;
float *dev_outputposition;
float *tmp_small_PoT; // Space for PoTs for reporting
float *tmp_PoT;
float *best_PoT;
float *tmp_PoT2;
float *best_PoT2;
hipDeviceProp_t gCudaDevProps;

int cudaAcc_init = 0;  // global count variable for CUDA mem allocations.


int cudaAcc_initialized() 
{
  return cudaAcc_init;
}


bool cudaAcc_setBlockingSync(int device) 
{
  //CUDA_ACC_SAFE_CALL(hipSetDeviceFlags(hipDeviceScheduleBlockingSync), false);
  CUDA_ACC_SAFE_CALL(hipSetDeviceFlags(hipDeviceScheduleYield), false);
  
  return true;
}


int cudaAcc_initializeDevice(int devPref, int usePolling) 
{
  int numCudaDevices = 0;
  int i = 0, bestDevFound =0;
  hipDeviceProp_t cDevProp[8];
  bool bCapableGPUFound = false;
  hipError_t cerr;
    
  // init our global DevProp var and query how many CUDA devices
  // are present.
  memset(&gCudaDevProps, 0, sizeof(hipDeviceProp_t));
  memset(cDevProp, 0, sizeof(hipDeviceProp_t)*8);
  
  //Jason: Don't use safecall with exit here, return 1 if an error finding devices etc.
  cerr = hipGetDeviceCount(&numCudaDevices);
  CUDA_ACC_SAFE_CALL_NO_SYNC("Couldn't get cuda device count\n");
  if(cerr != hipSuccess)  //Jason; Extra paranoia
    {
      fprintf(stderr, "setiathome_CUDA: hipGetDeviceCount() call failed.\n");
    }
  
  if(!numCudaDevices)
    {
      fprintf(stderr, "setiathome_CUDA: No CUDA devices found\n");
    }
  
  //limit to 16 GPU's for now
  if(numCudaDevices > 16) numCudaDevices = 16;
  
  fprintf(stderr, "setiathome_CUDA: Found %d CUDA device(s):\n", numCudaDevices);
  
  // Let's enumerate the CUDA devices avail and 
  // pick the best one.
  for(i = 0; i < numCudaDevices; i++)
    {
      CUDA_ACC_SAFE_CALL(hipGetDeviceProperties(&cDevProp[i], i), true);
#ifdef _WIN32
      fprintf(stderr, "  Device %d: %s, %u MiB, ",
	      i+1,
	      cDevProp[i].name,
	      (ULONGLONG)(cDevProp[i].totalGlobalMem>>20));
#else
      fprintf(stderr, "  Device %d: %s, %zu MiB, ",
	      i+1,
	      cDevProp[i].name,
	      (size_t)(cDevProp[i].totalGlobalMem>>20));
#endif	
      fprintf(stderr, "regsPerBlock %u\n",cDevProp[i].regsPerBlock);
	fprintf(stderr, "     computeCap %d.%d, multiProcs %d \n", 	
		cDevProp[i].major, cDevProp[i].minor,
		cDevProp[i].multiProcessorCount);
	//fprintf(stderr, "           totalGlobalMem = %d \n", cDevProp[i].totalGlobalMem);
	//fprintf(stderr, "           sharedMemPerBlock = %d \n",cDevProp[i].sharedMemPerBlock);
	//fprintf(stderr, "           regsPerBlock = %d \n", cDevProp[i].regsPerBlock);
        //fprintf(stderr, "           warpSize = %d \n",cDevProp[i].warpSize);
        //fprintf(stderr, "           memPitch = %d \n",cDevProp[i].memPitch);
        //fprintf(stderr, "           maxThreadsPerBlock = %d \n",cDevProp[i].maxThreadsPerBlock);
#if CUDART_VERSION >= 3000
	fprintf(stderr, "     pciBusID = %d, pciSlotID = %d\n", cDevProp[i].pciBusID, cDevProp[i].pciDeviceID);
#endif
	if(cDevProp[i].major < 3)
	  {
	    //Pre Kepler GPU, Cuda Runtime should report the clock rate correctly.
	    fprintf(stderr, "     clockRate = %d MHz\n", cDevProp[i].clockRate/1000);
	  } 
	//fprintf(stderr, "           totalConstMem = %d \n",cDevProp[i].totalConstMem);
	//fprintf(stderr, "           major = %d \n",cDevProp[i].major);
	//fprintf(stderr, "           minor = %d \n",cDevProp[i].minor);
	//fprintf(stderr, "           textureAlignment = %d \n",cDevProp[i].textureAlignment);
        //fprintf(stderr, "           deviceOverlap = %d \n",cDevProp[i].deviceOverlap);
        //fprintf(stderr, "           multiProcessorCount = %d \n",cDevProp[i].multiProcessorCount);
    }
  //nvFreeAPI();
  
  for(i = 0; i < numCudaDevices; i++)
    {
#if CUDART_VERSION >= 6050
      // Check the supported major revision to ensure it's valid and not some pre-Fermi
      if((cDevProp[i].major < 2))
	{
	  fprintf(stderr, "setiathome_CUDA: device %d is Pre-Fermi CUDA 2.x compute compatibility, only has %d.%d\n", 
		  i+1, cDevProp[i].major, cDevProp[i].minor);
	  continue;
	}
#else
      // Check the supported major revision to ensure it's valid and not emulation mode
      if((cDevProp[i].major < 1))
	{
	  fprintf(stderr, "setiathome_CUDA: device %d does not support CUDA 1.x compute compatibility, supports %d.%d\n", 
		  i+1, cDevProp[i].major, cDevProp[i].minor);
	  continue;
	  }
#endif
      
      // Check the supported major revision to ensure it's valid and not emulation mode
      if((cDevProp[i].major >= 9999))
	{
	  fprintf(stderr, "setiathome_CUDA: device %d is emulation device and should not be used, supports %d.%d\n", 
		  i+1, cDevProp[i].major, cDevProp[i].minor);
          continue;
	}
      
#if CUDART_VERSION < 3000
#pragma message (">>>PRE_FERMI_ONLY<<< Build\n")
      // Check the supported major revision for Pre-Fermi only Cuda 2.2 & 2.3 builds
      if((cDevProp[i].major > 1))
	{
	  fprintf(stderr, "setiathome_CUDA: device %d, compute capability %d.%d is not supported by this application\n", 
		  i+1, cDevProp[i].major, cDevProp[i].minor);
	  continue;
	}
#endif
      //Check if there is enough memory resources to handle our CUDA version of SETI
      if(cDevProp[i].totalGlobalMem < 128*1024*1024)
	  {
            fprintf(stderr, "setiathome_CUDA: device %d not have enough available global memory. Only found %d\n",
                    i+1, (int)cDevProp[i].totalGlobalMem);
            continue;
	  }
      
      //Check if this is a more powerful GPU than any others we found
      if(cDevProp[i].multiProcessorCount > gCudaDevProps.multiProcessorCount)
	{
	  memcpy(&gCudaDevProps, &cDevProp[i], sizeof(hipDeviceProp_t));
	  bestDevFound = i;
	  bCapableGPUFound = true;
	}
    }
  
  
  if(!devPref)
    {
      //fprintf(stderr,"In cudaAcc_initializeDevice(): Boinc passed DevPref %d, Which is choose best device %d\n",devPref,bestDevFound);
      
      if(bCapableGPUFound)
	{
	  fprintf(stderr, "setiathome_CUDA: No device specified, determined to use CUDA device %d: %s\n", bestDevFound+1, (char *)&cDevProp[bestDevFound].name);
	  CUDA_ACC_SAFE_CALL(hipSetDevice(bestDevFound),true);
	  if(!usePolling)
	    cudaAcc_setBlockingSync(bestDevFound);
	}
      else
	{
	  fprintf(stderr, "setiathome_CUDA: No SETI@home capabale CUDA GPU found...\n");
	  return 0;
	}
    }
  else 
    {
      fprintf(stderr,"In cudaAcc_initializeDevice(): Boinc passed DevPref %d\n",devPref);
      
      fprintf(stderr, "setiathome_CUDA: CUDA Device %d specified, checking...\n", devPref);
      // user must want a specific device, check it's qualifications
      if((devPref <= numCudaDevices)                             // Make sure it's a valid device
	 && (cDevProp[devPref-1].major >= 1)                     // that has at least 1.x compute
#if CUDART_VERSION < 3000
#pragma message (">>>PRE_FERMI_ONLY<<< Build\n")
	 // Check the supported major revision for Pre-Fermi only Cuda 2.2 & 2.3 builds
	 && (cDevProp[devPref-1].major < 2)
#endif
	 && (cDevProp[devPref-1].totalGlobalMem > 128*1024*1024) // and more than 128MB of memeory
	 && (cDevProp[devPref-1].major != 9999))                         // and is not an emulation device
	{
	  fprintf(stderr, "   Device %d: %s is okay\n", devPref, (char *)&cDevProp[devPref-1].name);
	  memcpy(&gCudaDevProps, &cDevProp[devPref - 1], sizeof(hipDeviceProp_t));
	  CUDA_ACC_SAFE_CALL(hipSetDevice(devPref - 1), true);
	  if(!usePolling)
	    cudaAcc_setBlockingSync(devPref - 1);                
	}
      else
	{
	  fprintf(stderr, "   Device cannot be used\n");
	  return 0;
	}
    }
  
#if(CUDART_VERSION >= 4000)
  // find_pulse_kernels are limited by memory bandwidth and suboptimal fetches.
  // Prefer a larger L1 cache, as we don't need all the shared memory offered (48k)
  // Override for specific kernels where we need the shared memory instead
  // (e.g. find_triplets_kernel)
  
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
#endif
  
  //  gpu device heuristics
  //if(gCudaDevProps.major >= 2) { cuda_tmax = 1024; cudaps_blksize = 256; }
  //else if(gCudaDevProps.minor == 3) { cuda_tmax = 512; cudaps_blksize = 128; }
  //else { cuda_tmax = 256; cudaps_blksize = 64; }
  
  //fprintf(stderr, "   Guru says: Max threads is %d/blk, best for Powerspectrum is %d/blk\n", cuda_tmax, cudaps_blksize);
  
  //fprintf(stderr,"-->In cudaAcc_initializeDevice(): 'Supposed' active Cuda device has %d multiProcessors.\n",gCudaDevProps.multiProcessorCount);
  //fprintf(stderr,"-->In cudaAcc_initializeDevice(): 'Supposed' active Cuda device has %d regsPerBlock.\n",gCudaDevProps.regsPerBlock);
  
  //cudaAcc_init_exit_proc();
  
  return 1;
}

//const double SPLITTER=(1<<BSPLIT)+1;
//__host__ float2 splitdd(double a) {
//    double t = a*SPLITTER; 
//	double ahi= t-(t-a);
//	double alo = a-ahi;
//
//	return make_float2((float)ahi,(float)alo);
//}

int cudaAcc_initialize(sah_complex* cx_DataArray, int NumDataPoints, int gauss_pot_length, unsigned long nsamples,
		       double gauss_null_chi_sq_thresh, double gauss_chi_sq_thresh,
		       double pulse_display_thresh, double PulseThresh, int PulseMax,					   
		       double sample_rate, long acfftlen)
{	
  hipError_t cu_err;
  
  //Prevent cudaAcc_initialize to be re-entrant
  if(cudaAcc_init)
    return 0;
  
  cu_err = hipMalloc((void**) &dev_cx_DataArray, sizeof(*dev_cx_DataArray) * (NumDataPoints*PADVAL));
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_cx_DataArray");
      return 1;
    } else { CUDAMEMPRINT(dev_cx_DataArray,"hipMalloc((void**) &dev_cx_DataArray",NumDataPoints,sizeof(*dev_cx_DataArray)); };
  cudaAcc_init++;
  CUDA_ACC_SAFE_CALL(hipMemcpyAsync(dev_cx_DataArray, cx_DataArray, NumDataPoints * sizeof(*cx_DataArray), hipMemcpyHostToDevice),true);
  //CUDA_ACC_SAFE_CALL((CUDASYNC),true);
  
  cu_err = hipMalloc((void**) &dev_cx_ChirpDataArray, sizeof(*dev_cx_ChirpDataArray) * (NumDataPoints*2*PADVAL));    
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_cx_ChirpDataArray");
      return 1;
    }  else { CUDAMEMPRINT(dev_cx_ChirpDataArray,"hipMalloc((void**) &dev_cx_ChirpDataArray",NumDataPoints*PADVAL,sizeof(*dev_cx_ChirpDataArray)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_flag, sizeof(*dev_flag));
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_flag");
      return 1;
    }  else { CUDAMEMPRINT(dev_flag,"hipMalloc((void**) &dev_flag",1,sizeof(*dev_flag)); };
  cudaAcc_init++;
  
  
  cu_err = hipMalloc((void**) &dev_WorkData, sizeof(*dev_WorkData) * NumDataPoints * PADVAL); // + 1/8 for find_pulse));
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_WorkData");
      return 1;
    } else { CUDAMEMPRINT(dev_WorkData,"hipMalloc((void**) &dev_WorkData",NumDataPoints * PADVAL,sizeof(*dev_WorkData)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_PowerSpectrum, sizeof(*dev_PowerSpectrum) * NumDataPoints * PADVAL);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_PowerSpectrum");
      return 1;
    } else { CUDAMEMPRINT(dev_PowerSpectrum,"hipMalloc((void**) &dev_PowerSpectrum",NumDataPoints,sizeof(*dev_PowerSpectrum)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_t_PowerSpectrum, sizeof(*dev_t_PowerSpectrum) * (NumDataPoints+8));
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_t_PowerSpectrum");
      return 1;
    } else { CUDAMEMPRINT(dev_t_PowerSpectrum,"hipMalloc((void**) &dev_t_PowerSpectrum",NumDataPoints+8,sizeof(*dev_t_PowerSpectrum)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_GaussFitResults, sizeof(*dev_GaussFitResults) * NumDataPoints);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_GaussFitResults");
      return 1;
    } else { CUDAMEMPRINT(dev_GaussFitResults,"hipMalloc((void**) &dev_GaussFitResults",NumDataPoints,sizeof(*dev_GaussFitResults)); };
  cudaAcc_init++;

  cu_err = hipMalloc((void**) &dev_TripletResults, sizeof(*dev_GaussFitResults) * NumDataPoints);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_TripletResults");
      return 1;
    } else { CUDAMEMPRINT(dev_TripletResults,"hipMalloc((void**) &dev_TripletResults",NumDataPoints,sizeof(*dev_TripletResults)); };
  cudaAcc_init++;

  cu_err = hipMalloc((void**) &dev_PulseResults, sizeof(*dev_PulseResults) * NumDataPoints);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_PulseResults");
      return 1;
    } else { CUDAMEMPRINT(dev_PulseResults,"hipMalloc((void**) &dev_PulseResults",NumDataPoints,sizeof(*dev_PulseResults)); };
  cudaAcc_init++;
  
  dev_GaussFitResultsReordered = dev_GaussFitResults + NumDataPoints;
  dev_GaussFitResultsReordered2 = dev_GaussFitResultsReordered + NumDataPoints;
  //CUDA_ACC_SAFE_CALL(hipMalloc((void**) &dev_GaussFitResultsReordered, sizeof(*dev_GaussFitResultsReordered) * NumDataPoints)); // TODO: it can be smaller
  //CUDA_ACC_SAFE_CALL(hipMalloc((void**) &dev_GaussFitResultsReordered2, sizeof(*dev_GaussFitResultsReordered2) * NumDataPoints)); // TODO: it can be smaller
  
#ifdef PINNED
  hipHostMalloc((void **)&GaussFitResults, sizeof(*GaussFitResults) * NumDataPoints);
  hipHostMalloc((void **)&TripletResults, sizeof(*GaussFitResults) * NumDataPoints);
  hipHostMalloc((void **)&PulseResults, sizeof(*GaussFitResults) * NumDataPoints);
#else
  GaussFitResults = (float4*) malloc(sizeof(*GaussFitResults) * NumDataPoints);
#endif
  GaussFitResults2 = GaussFitResults + NumDataPoints;
  
  cu_err = hipMalloc((void**) &dev_PoT, sizeof(*dev_PoT) * NumDataPoints * PADVAL_PULSE); // + 1/2 for find_pulse
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_PoT");
      return 1;
    } else { CUDAMEMPRINT(dev_PoT,"hipMalloc((void**) &dev_PoT",NumDataPoints * PADVAL_PULSE,sizeof(*dev_PoT)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_PoTPrefixSum, sizeof(*dev_PoTPrefixSum) * NumDataPoints * PADVAL_PULSE); // + 1/2 for find_pulse
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_PoTPrefixSum");
      return 1;
    } else { CUDAMEMPRINT(dev_PoTPrefixSum,"hipMalloc((void**) &dev_PoTPrefixSum", NumDataPoints * PADVAL_PULSE,sizeof(*dev_PoTPrefixSum)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_NormMaxPower, sizeof(*dev_NormMaxPower) * NumDataPoints / gauss_pot_length);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_NormMaxPower");
      return 1;
    } else { CUDAMEMPRINT(dev_NormMaxPower,"hipMalloc((void**) &dev_NormMaxPower", NumDataPoints / gauss_pot_length,sizeof(*dev_NormMaxPower)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_flagged, sizeof(*dev_flagged) * NumDataPoints);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_flagged");
      return 1;
    } else { CUDAMEMPRINT(dev_flagged,"hipMalloc((void**) &dev_flagged",NumDataPoints,sizeof(*dev_flagged)); };
  cudaAcc_init++;
  
  cu_err = hipMalloc((void**) &dev_outputposition, sizeof(*dev_outputposition) * NumDataPoints);
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_outputposition");
      return 1;
    } else { CUDAMEMPRINT(dev_outputposition,"hipMalloc((void**) &dev_outputposition",NumDataPoints,sizeof(*dev_outputposition)); };
  cudaAcc_init++;
  
  dev_best_pot = (float*) dev_WorkData;
  dev_report_pot = dev_PoT;
#ifdef PINNED
  hipHostMalloc((void **)&tmp_PoT, NumDataPoints * sizeof(*tmp_PoT) * 3 / 2);
  hipHostMalloc((void **)&best_PoT, NumDataPoints * sizeof(*best_PoT) * 3 / 2);
  hipHostMalloc((void **)&tmp_PoT2, NumDataPoints * sizeof(*tmp_PoT) * 3 / 2);
  hipHostMalloc((void **)&best_PoT2, NumDataPoints * sizeof(*best_PoT) * 3 / 2);
#else
  tmp_PoT = (float*) malloc(NumDataPoints * sizeof(*tmp_PoT) * 3 / 2);
  best_PoT = (float*) malloc(NumDataPoints * sizeof(*best_PoT) * 3 / 2);
#endif
  
  CUDA_ACC_SAFE_CALL( (cu_err = hipMalloc((void**) &dev_PowerSpectrumSumMax, sizeof(*dev_PowerSpectrumSumMax) * NumDataPoints*2 / 8)),true); // The ffts are at least 8 elems long
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_PowerSpectrumSumMax");
      return 1;
    } else { CUDAMEMPRINT(dev_PowerSpectrumSumMax,"hipMalloc((void**) &dev_PowerSpectrumSumMax",NumDataPoints*2 / 8,sizeof(*dev_PowerSpectrumSumMax)); };
  cudaAcc_init++;
  
  dev_tmp_pot = (float*) dev_PoTPrefixSum; // next do pot2

  cu_err = hipMalloc((void**) &dev_tmp_pot2, sizeof(*dev_PoTPrefixSum) * NumDataPoints * PADVAL_PULSE); // + 1/2 for find_pulse
  if(hipSuccess != cu_err) 
    {
      CUDA_ACC_SAFE_CALL_NO_SYNC("hipMalloc((void**) &dev_tmp_pot2");
      return 1;
    } else { CUDAMEMPRINT(dev_tmp2_pot_2,"hipMalloc((void**) &dev_tmp_po2", NumDataPoints * PADVAL_PULSE,sizeof(*dev_PoTPrefixSum)); };
  cudaAcc_init++;
  
  cudaAcc_NumDataPoints = NumDataPoints;
#ifdef PINNED
  hipHostMalloc((void **)&tmp_small_PoT, NumDataPoints / 8 * sizeof(*tmp_small_PoT));
  hipHostMalloc((void **)&PowerSpectrumSumMax, sizeof(*dev_PowerSpectrumSumMax) * NumDataPoints / 8);		
#else
  tmp_small_PoT = (float*) malloc(NumDataPoints / 8 * sizeof(*tmp_small_PoT));
  PowerSpectrumSumMax = (float3*) malloc(sizeof(*dev_PowerSpectrumSumMax) * NumDataPoints / 8);		
#endif
  
  cu_err = hipStreamCreate(&cudaAutocorrStream);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 0 failed\r\n"); return 1;}
/*  cu_err = hipStreamCreate(&cudaAutocorrStream[1]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 1 failed\r\n"); return 1;}
  cu_err = hipStreamCreate(&cudaAutocorrStream[2]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 2 failed\r\n"); return 1;}
  cu_err = hipStreamCreate(&cudaAutocorrStream[3]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 3 failed\r\n"); return 1;}
  cu_err = hipStreamCreate(&cudaAutocorrStream[4]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 4 failed\r\n"); return 1;}
  cu_err = hipStreamCreate(&cudaAutocorrStream[5]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 5 failed\r\n"); return 1;} 
  cu_err = hipStreamCreate(&cudaAutocorrStream[6]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 6 failed\r\n"); return 1;}
  cu_err = hipStreamCreate(&cudaAutocorrStream[7]);
  if(hipSuccess != cu_err) 
    {fprintf(stderr, "Autocorr stream create 7 failed\r\n"); return 1;}
*/
  
  hipStreamCreate(&fftstream1);

  if(cudaAcc_initializeGaussfit(PoTInfo, gauss_pot_length, nsamples, gauss_null_chi_sq_thresh, gauss_chi_sq_thresh))
    {
      fprintf(stderr, "GaussFit Init failed...\n");
      return 1;
    }    
  
  if(cudaAcc_initialize_pulse_find(pulse_display_thresh, PulseThresh, PulseMax))
    {
      fprintf(stderr, "PulseFind Init failed...\n");
      return 1;		
    }
  
#ifdef _WIN32
#if CUDART_VERSION >= 3000
  if(gCudaDevProps.major >= 3)
    {
      //Kepler GPU, has complex clock setup, dig into nvapi on Windows.
      int crate = nvGetCurrentClock( gCudaDevProps.pciBusID, gCudaDevProps.pciDeviceID);		
      if(crate)
	fprintf(stderr, "\nGPU current clockRate = %d MHz\n\n",crate/1000);
      nvFreeAPI();
    }
#endif
#endif //_WIN32
  
  if(acfftlen && cudaAcc_InitializeAutocorrelation(acfftlen))
    {
      fprintf(stderr, "Not enough VRAM for Autocorrelations...\n");
      return 1;
    }
  
  //...All good
#if CUDART_VERSION >= 3000
  if(gCudaDevProps.major >= 2)
    {
      size_t threadlimit = 0;
      cudaThreadGetLimit(&threadlimit,hipLimitStackSize);
      fprintf(stderr,"Thread call stack limit is: %dk\n", (int)threadlimit/1024);
      //if(threadlimit < 10240)
      //{
      //	hipError_t cerr =  cudaThreadSetLimit(hipLimitStackSize, 10240);
      //	if(cerr != hipSuccess)
      //	{
      //		fprintf(stderr,"CudaThreadSetLimit() returned code %s\n", cerr);
      //	}
      //	else
      //	{
      //		threadlimit = 0;
      //		cudaThreadGetLimit(&threadlimit,hipLimitStackSize);
      //		fprintf(stderr,"Cuda Thread Limit was adjusted to %dk\n", threadlimit/1024);
      //	}
      //}
    }
#endif
  
  CUDASYNC;  //clear any error codes.
  
  return 0;
}


int cudaAcc_InitializeAutocorrelation(int ac_fftlen)
{
  //    hipError_t cu_err;
  hipfftResult cu_errf;
  // Failure to initialise Cuda device memory for Autocorrelation isn't fatal, but we need to keep track of things...
  gCudaAutocorrelation = (ac_fftlen > 0); // initially assume we're going to do Autocorrelations on GPU if needed
  dev_AutoCorrIn[0] = NULL;
  dev_AutoCorrOut[0] = NULL;
  cudaAutoCorr_plan = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
/*  cudaAutoCorr_plan[1] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[2] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[3] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[4] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[5] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[6] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[7] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
*/
  int ac_size = sizeof(*dev_AutoCorrIn[0])*ac_fftlen*4;
//  int ac_sizeR = sizeof(*dev_AutoCorrIn)*ac_fftlen*4;
  
  if(gCudaAutocorrelation)
    {
      dev_AutoCorrIn[0] = (float2 *) dev_GaussFitResults;
      hipMalloc((void **)&dev_AutoCorrIn[1], ac_size);
      hipMalloc((void **)&dev_AutoCorrIn[2], ac_size);
      hipMalloc((void **)&dev_AutoCorrIn[3], ac_size);
      hipMalloc((void **)&dev_AutoCorrIn[4], ac_size);
      hipMalloc((void **)&dev_AutoCorrIn[5], ac_size);
      hipMalloc((void **)&dev_AutoCorrIn[6], ac_size);
      hipMalloc((void **)&dev_AutoCorrIn[7], ac_size);
      fprintf(stderr,"re-using dev_GaussFitResults array for dev_AutoCorrIn, %d bytes\n",ac_size);
    }
  
  if(gCudaAutocorrelation)
    {
      dev_AutoCorrOut[0] = &dev_AutoCorrIn[0][ac_fftlen*4];
      hipMalloc((void **)&dev_AutoCorrOut[1], ac_size);
      hipMalloc((void **)&dev_AutoCorrOut[2], ac_size);
      hipMalloc((void **)&dev_AutoCorrOut[3], ac_size);
      hipMalloc((void **)&dev_AutoCorrOut[4], ac_size);
      hipMalloc((void **)&dev_AutoCorrOut[5], ac_size);
      hipMalloc((void **)&dev_AutoCorrOut[6], ac_size);
      hipMalloc((void **)&dev_AutoCorrOut[7], ac_size);
      fprintf(stderr,"re-using dev_GaussFitResults+%dx%d array for dev_AutoCorrOut, %d bytes\n",ac_fftlen*4,(int)sizeof(*dev_AutoCorrOut),ac_size);
    }
  
  if(gCudaAutocorrelation)
    {
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan, ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 0(4NFFT method)\n");
/*      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[1], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 1(4NFFT method)\n");
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[2], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 2(4NFFT method)\n");
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[3], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 3(4NFFT method)\n");
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[4], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 4(4NFFT method)\n");
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[5], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 5(4NFFT method)\n");
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[6], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 6(4NFFT method)\n");
      cu_errf = hipfftPlan1d(&cudaAutoCorr_plan[7], ac_fftlen*4, HIPFFT_C2C, 1); //4N FFT method
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr,"Not enough room for autocorrelation CuFFT plan 7(4NFFT method)\n");
*/
/*      cu_errf = hipfftSetStream(cudaAutoCorr_plan[0], cudaAutocorrStream[0]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 0 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[1], cudaAutocorrStream[1]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 1 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[2], cudaAutocorrStream[2]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 2 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[3], cudaAutocorrStream[3]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 3 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[4], cudaAutocorrStream[4]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 4 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[5], cudaAutocorrStream[5]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 5 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[6], cudaAutocorrStream[6]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 6 failed");
      cu_errf = hipfftSetStream(cudaAutoCorr_plan[7], cudaAutocorrStream[7]);
      if(HIPFFT_SUCCESS != cu_errf) 
        fprintf(stderr, "hipfftSetStream 7 failed");
*/
//      cu_errf = hipfftPlan1d(&cudaAutoCorr_planR, ac_fftlen*2, HIPFFT_R2C, 1); //4N FFT method
      
      if(HIPFFT_SUCCESS != cu_errf) 
	{
	  fprintf(stderr,"Not enough room for autocorrelation CuFFT plan (4NFFT method)\n");
	  //These aren't allocated anymore, but re-use other areas
	  //hipFree(dev_AutoCorrOut);  // If we can't do the fft, won't be needing the output either.
	  //hipFree(dev_AutoCorrIn);  // If we can't do the output, won;t be needing the input either.
	  gCudaAutocorrelation = false;
	  dev_AutoCorrIn[0] = NULL;
	  dev_AutoCorrOut[0] = NULL;
	  cudaAutoCorr_plan = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
	  return 1;
	}
#if CUDART_VERSION >= 3000
      cufftSetCompatibilityMode(cudaAutoCorr_plan, CUFFT_COMPATIBILITY_NATIVE);
/*      cufftSetCompatibilityMode(cudaAutoCorr_plan[1],CUFFT_COMPATIBILITY_NATIVE);
      cufftSetCompatibilityMode(cudaAutoCorr_plan[2],CUFFT_COMPATIBILITY_NATIVE);
      cufftSetCompatibilityMode(cudaAutoCorr_plan[3],CUFFT_COMPATIBILITY_NATIVE);
      cufftSetCompatibilityMode(cudaAutoCorr_plan[4],CUFFT_COMPATIBILITY_NATIVE);
      cufftSetCompatibilityMode(cudaAutoCorr_plan[5],CUFFT_COMPATIBILITY_NATIVE);
      cufftSetCompatibilityMode(cudaAutoCorr_plan[6],CUFFT_COMPATIBILITY_NATIVE);
      cufftSetCompatibilityMode(cudaAutoCorr_plan[7],CUFFT_COMPATIBILITY_NATIVE);*/
      
#endif
    }
  
  dev_ac_partials[0] = (float3 *) dev_AutoCorrOut[0];
  dev_ac_partials[1] = (float3 *) dev_AutoCorrOut[1];
  dev_ac_partials[2] = (float3 *) dev_AutoCorrOut[2];
  dev_ac_partials[3] = (float3 *) dev_AutoCorrOut[3];
  dev_ac_partials[4] = (float3 *) dev_AutoCorrOut[4];
  dev_ac_partials[5] = (float3 *) dev_AutoCorrOut[5];
  dev_ac_partials[6] = (float3 *) dev_AutoCorrOut[6];
  dev_ac_partials[7] = (float3 *) dev_AutoCorrOut[7];

  hipHostMalloc((void **)&blockSums[0], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[1], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[2], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[3], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[4], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[5], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[6], 1024*sizeof(float3));
  hipHostMalloc((void **)&blockSums[7], 1024*sizeof(float3));
  
  return 0;
}


void cudaAcc_free_AutoCorrelation()
{
  if(cudaAutoCorr_plan) hipfftDestroy(cudaAutoCorr_plan);
/*  if(cudaAutoCorr_plan[1]) hipfftDestroy(cudaAutoCorr_plan[1]);
  if(cudaAutoCorr_plan[2]) hipfftDestroy(cudaAutoCorr_plan[2]);
  if(cudaAutoCorr_plan[3]) hipfftDestroy(cudaAutoCorr_plan[3]);
  if(cudaAutoCorr_plan[4]) hipfftDestroy(cudaAutoCorr_plan[4]);
  if(cudaAutoCorr_plan[5]) hipfftDestroy(cudaAutoCorr_plan[5]);
  if(cudaAutoCorr_plan[6]) hipfftDestroy(cudaAutoCorr_plan[6]);
  if(cudaAutoCorr_plan[7]) hipfftDestroy(cudaAutoCorr_plan[7]);
*/
  //These aren't allocated anymore, but re-use other areas
  if(blockSums[0]) hipHostFree(blockSums[0]);
  if(blockSums[1]) hipHostFree(blockSums[1]);
  if(blockSums[2]) hipHostFree(blockSums[2]);
  if(blockSums[3]) hipHostFree(blockSums[3]);
  if(blockSums[4]) hipHostFree(blockSums[4]);
  if(blockSums[5]) hipHostFree(blockSums[5]);
  if(blockSums[6]) hipHostFree(blockSums[6]);
  if(blockSums[7]) hipHostFree(blockSums[7]);
#pragma message("You should free autocorr extra pointers 1-7");
  gCudaAutocorrelation = false;
  dev_AutoCorrIn[0] = NULL;
  dev_AutoCorrOut[0] = NULL;
  cudaAutoCorr_plan = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
/*  cudaAutoCorr_plan[1] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[2] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[3] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[4] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[5] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[6] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
  cudaAutoCorr_plan[7] = 0; // hipfftHandle is not a pointer. Cannot be set to "NULL"
*/
  return;
}

#define CF(_ptr) do { hipFree(_ptr);_ptr = NULL; } while (0);


void cudaAcc_free() {
  fprintf(stderr,"cudaAcc_free() called...\n");
  if(!cudaAcc_init) return;
  fprintf(stderr,"cudaAcc_free() running...\n");
  cudaAcc_free_pulse_find();
  fprintf(stderr,"cudaAcc_free() PulseFind freed...\n");
  cudaAcc_free_Gaussfit();
  fprintf(stderr,"cudaAcc_free() Gaussfit freed...\n");
  cudaAcc_free_AutoCorrelation();
  fprintf(stderr,"cudaAcc_free() AutoCorrelation freed...\n");
  
  //hipStreamDestroy(fftstream1);
  //hipStreamDestroy(fftstream0);
  
  switch(cudaAcc_init) 
    {
    case 16:
      CF(dev_PulseResults);
    case 15:
      CF(dev_TripletResults);
    case 14:
      CF(dev_tmp_pot2);
    case 13:
      CF(dev_PowerSpectrumSumMax);
    case 12:
      CF(dev_outputposition);
    case 11:
      CF(dev_flagged);
    case 10:
      CF(dev_NormMaxPower);
    case 9:
      CF(dev_PoTPrefixSum);
    case 8:
      CF(dev_PoT);
    case 7:
      CF(dev_GaussFitResults);
    case 6:
      CF(dev_t_PowerSpectrum);
    case 5:
      CF(dev_PowerSpectrum);
    case 4:
      CF(dev_WorkData);
    case 3:
      CF(dev_flag);
    case 2:
      CF(dev_cx_ChirpDataArray);
    case 1:
      CF(dev_cx_DataArray);
    case 0:
    default:
      //CUDA_ACC_SAFE_CALL(CF(dev_GaussFitResultsReordered));
      //CUDA_ACC_SAFE_CALL(CF(dev_GaussFitResultsReordered2));
      //cudaAcc_deallocBlockSums(); // scans are not used at the moment
#ifdef PINNED
      hipHostFree(GaussFitResults);
      hipHostFree(TripletResults);
      hipHostFree(PulseResults);
      hipHostFree(tmp_small_PoT);
      hipHostFree(tmp_PoT);
      hipHostFree(best_PoT);
      hipHostFree(tmp_PoT2);
      hipHostFree(best_PoT2);
#else
      free(GaussFitResults);	
      free(tmp_small_PoT);
#endif
    }
  
  cudaAcc_fft_free();
  cudaAcc_init = 0;
  cmem_rtotal = 0;
#if(CUDART_VERSION >= 4000)
  hipDeviceReset();
#else
  hipDeviceReset();
#endif
  fprintf(stderr,"cudaAcc_free() DONE.\n");
}
#endif
