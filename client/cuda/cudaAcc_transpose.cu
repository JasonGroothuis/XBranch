#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#include "cudaAcceleration.h"
#ifdef USE_CUDA

#include "cudaAcc_data.h"
#include "cudaAcc_utilities.h"

#define BLOCK_DIM 16

// Transpose function from sample in NVIDIA CUDA SDK
__global__ void cudaAcc_transpose(float *odata, float *idata, int width, int height)
{
	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];

	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

void cudaAcc_transposeGPU(float *odata, float *idata, int width, int height) {
	if (!cudaAcc_initialized()) return;

	dim3 block(BLOCK_DIM, BLOCK_DIM, 1);
	dim3 grid((width + BLOCK_DIM - 1) / BLOCK_DIM, (height + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	// no need to copy data from host, this data is already on device thanks to cudaAcc_execute_dfts
	cudaAcc_transpose<<<grid, block>>>(odata, idata, width, height);
	CUDA_ACC_SAFE_CALL_NO_SYNC("cudaAcc_transpose");
}
#endif //USE_CUDA
