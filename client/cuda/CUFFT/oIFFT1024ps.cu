#include "hip/hip_runtime.h"
#include "codelets.h"

__global__ void oIFFT1024_device_ps( float *ps, float2 *src )
{	
    int tid = threadIdx.x;
    
    int iblock = blockIdx.y * gridDim.x + blockIdx.x;
    int index = iblock * 1024 + tid;
    src += index;
    ps += index;
    
    int hi4 = tid>>4;
    int lo4 = tid&15;
    int hi2 = tid>>4;
    int mi2 = (tid>>2)&3;
    int lo2 = tid&3;

    float2 a[16];
    __shared__ float smem[69*16];
    
    load<16>( a, src, 64 );

    IFFT16( a );
    
    itwiddle<16>( a, tid, 1024 );
    int il[] = {0,1,2,3, 16,17,18,19, 32,33,34,35, 48,49,50,51};
    transpose<16>( a, &smem[lo4*65+hi4], 4, &smem[lo4*65+hi4*4], il );
    
    IFFT4x4( a );

    itwiddle4x4( a, lo4 );
    transpose4x4( a, &smem[hi2*17 + mi2*4 + lo2], 69, &smem[mi2*69*4 + hi2*69 + lo2*17 ], 1, 0xE );
    
    IFFT16( a );

    //store<16>( a, dst, 64 );
	#pragma unroll
    for( int i = 0; i < 16; i++ )
	{
		 float2 freqData = a[rev<16>(i)];
		// PowerSpectrum[i] = freqData.x * freqData.x + freqData.y * freqData.y;
        ps[i*64] = freqData.x * freqData.x + freqData.y * freqData.y;
	}
}   
    
extern "C" void oIFFT1024ps( float *ps, float2 *src, int batch )
{	
    oIFFT1024_device_ps<<< grid2D(batch), 64 >>>( ps, src );
}	
