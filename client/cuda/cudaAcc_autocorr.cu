#include "hip/hip_runtime.h"

#include "cudaAcceleration.h"

#ifdef USE_CUDA

#include "cudaAcc_data.h"
#include "cudaAcc_utilities.h"

//#define B 2


#define RPI 64
#define RPIB 8
#define RPBY 4

#define B 8
#define RPS 64
#define RDP 256

float2 *dev_AutoCorrIn[8];
float2 *dev_AutoCorrOut[8];


bool gCudaAutocorrelation = false;

float3 *dev_blockSums[8];
float3 *blockSums[8];
float3 *dev_ac_partials[8];

//float ac_TotalSum;
//float ac_Peak;
//int ac_PeakBin;



__global__ void __launch_bounds__(RPI, 16)
ac_RepackInputKernelR(float *PowerSpectrum, float2 *dct_In) 
{
  int sidx = (threadIdx.x + blockIdx.x*RPI + (blockIdx.y*RPI*RPIB)); 
  int nDestPoints = 524288/4; // (gridDim.y*RPI*RPIB)*4;
  int didx1 = sidx<<2; 
  int didx2 = ((nDestPoints)<<3)-didx1-4; 

  PowerSpectrum += sidx;
  float *dct1 = (float*)((void*)dct_In + didx1);
  float *dct2 = (float*)((void*)dct_In + didx2);
  //  float2 t = make_float2(PowerSpectrum[0], 0);
  float t = PowerSpectrum[0];
      
  *dct1 = t;
  *dct2 = t;
}



__global__ void __launch_bounds__(RPI, 16)
ac_RepackInputKernelP(float *PowerSpectrum, float2 *dct_In) 
{
  int sidx = (threadIdx.x + blockIdx.x*RPI + (blockIdx.y*RPI*RPIB)); 
  int nDestPoints = 524288; // (gridDim.y*RPI*RPIB)*4;
  int didx1 = sidx<<4; 
  int didx2 = ((nDestPoints-2)<<3)-didx1; 

  PowerSpectrum += sidx;
  float4 *dct1 = (float4*)((void*)dct_In + didx1);
  float4 *dct2 = (float4*)((void*)dct_In + didx2);
  float4 t = make_float4(0.0f, 0.0f, PowerSpectrum[0], 0.0f);
      
  *dct1 = t;
  *dct2 = t;
}


__global__ void __launch_bounds__(RPI, 16)
ac_RepackInputKernel(float *PowerSpectrum, float2 *dct_In) 
{
  int sidx = (threadIdx.x + (blockIdx.x*RPI*RPIB)); 
  int nDestPoints = 524288; //(gridDim.x*RPI*RPIB)<<2;
  int didx1 = sidx<<4; 
  int didx2 = ((nDestPoints-2)<<3)-didx1 - 2*(RPIB-1)*RPI<<3; 
  
  float a[RPIB];
  PowerSpectrum += sidx;
  float2 *dct1 = (float2*)((void*)dct_In + didx1);
  float2 *dct2 = (float2*)((void*)dct_In + didx2);

#pragma unroll
  for(int i = 0; i < RPIB; i++)
    {
      a[i] = PowerSpectrum[i*RPI];
    }
  
#pragma unroll
  for(int i = 0; i < RPIB; i++)
    {
      float4 t = make_float4(0.0f, 0.0f, a[i], 0.0f);
      float4 t2 = make_float4(0.0f, 0.0f, a[RPIB-1-i], 0.0f);
      
      *(float4 *)(&dct1[2*i*RPI]) = t;
      *(float4 *)(&dct2[2*i*RPI]) = t2;
    }
}


/*

__global__ void 
ac_RepackInputKernel(float * __restrict__ PowerSpectrum, float2 * __restrict__ dct_In) 
{
  int sidx = (threadIdx.x + (blockIdx.x*RPI*RPIB)); 
  int nDestPoints = (gridDim.x*RPI*RPIB)<<2;
  int didx1 = sidx<<1; 
  int didx2 = (nDestPoints-2)-didx1; 
  
  float a[RPIB];
  PowerSpectrum += sidx;
  float2 * __restrict__ dct1 = (float2*)((void*)dct_In + sizeof(float2)*didx1);
  float2 * __restrict__ dct2 = (float2*)((void*)dct_In + sizeof(float2)*didx2);

#pragma unroll
  for(int i = 0; i < RPIB; i++)
    {
      a[i] = PowerSpectrum[i*RPI];
    }
  
#pragma unroll
  for(int i = 0; i < RPIB; i++)
    {
      float4 t = make_float4(0.0f, 0.0f, a[i], 0.0f);
      
      *(float4 *)(&dct1[2*i*RPI]) = t;
      *(float4 *)(&dct2[-(2*i*RPI)]) = t;
    }
}
*/



__global__ void __launch_bounds__(RPS, 16)
  ac_RepackScaleKernelP(float2 *src, float2 *dst) 
{
  int didx = ((threadIdx.x + blockIdx.x*RPS + blockIdx.y*RPS*B));  //packing into float2s
  int sidx = didx << 1; //((threadIdx.x + blockIdx.x*RPS*B)*2);

//printf("bx=%d, by=%d, tx=%d, sidx=%d\r\n", blockIdx.x, blockIdx.y, threadIdx.x, sidx);
  dst += didx;
  src += sidx;

  float4 t = ((float4 *)src)[0];
  float2 a = make_float2(t.x, t.z);

  a.x *= a.x;
  a.y *= a.y;

  dst[0] = a;
}

__global__ void __launch_bounds__(RPS, 16)
ac_RepackScaleKernel(float2 *src, float2 *dst) 
{
  int didx = ((threadIdx.x + blockIdx.x*RPS*B));  //packing into float2s
  int sidx = didx << 1; //((threadIdx.x + blockIdx.x*RPS*B)*2);
  
  float2 a[B];
  
  dst += didx;
  src += sidx;

#pragma unroll
  for(int i = 0; i < B; i++)
    {
      float4 t = *(float4 *)(&src[i*RPS*2]);
      a[i] = make_float2(t.x, t.z);
    }
  for(int i = 0; i < B; i++)
    {
      a[i].x *= a[i].x;
      a[i].y *= a[i].y;
    }
#pragma unroll
  for(int i = 0; i < B; i++)
    {
      dst[i*RPS] = a[i];
    }
}

extern __shared__ float3 acpartial[];

__global__ void ac_reducePartial(float *ac, float3 *devpartials)
{
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockIdx.x*RDP;
  const int bid = blockIdx.x;
  int n = RDP>>1;
  
  float3 *acp = acpartial;
  float tmp = ac[idx];
  acp[tid].z = idx;
  acp[tid].y = idx >= 1 ? tmp : 0.0f; 
  acp[tid].x = tmp;
  
  __syncthreads();
  
  volatile float3 *dp = &acp[tid];
  int fadd = n * 12;
#pragma unroll 2
  for(; n > 32; n >>= 1)
    {
      if(tid < n)
	{
	  float a =  __fadd_rn(dp[0].x, (*(float3*)(((void*)dp)+fadd)).x);
	  // peak power & its bin
	  float pp = (*(float3*)(((void*)dp)+fadd)).y;
	  float pb = (*(float3*)(((void*)dp)+fadd)).z;
	  fadd >>= 1;
	  bool b = pp > dp[0].y;
          dp[0].x = a;
	  if(b)
	    {
	      dp[0].y = pp;
	      dp[0].z = pb;
	    }
	}
      __syncthreads();
    }

#pragma unroll 6
  for(; n > 0; n >>= 1)
    {
      if(tid < n)
	{
	  float a =  __fadd_rn(dp[0].x, (*(float3*)(((void*)dp)+fadd)).x);
	  // peak power & its bin
	  float pp = (*(float3*)(((void*)dp)+fadd)).y;
	  float pb = (*(float3*)(((void*)dp)+fadd)).z;
	  fadd >>= 1;
	  bool b = pp > dp[0].y;
          dp[0].x = a;
	  if(b)
	    {
	      dp[0].y = pp;
	      dp[0].z = pb;
	    }
	}
    }

  if(tid == 0) 
    {        
      devpartials[bid] = make_float3(dp[0].x, dp[0].y, dp[0].z);
    }
}



int cudaAcc_FindAutoCorrelations(float *AutoCorrelation, int ac_fftlen) 
{
  for(int fft_num = 0; fft_num < 8; fft_num++)
    {
      //Jason: Use 4N-FFT method for Type 2 Discrete Cosine Tranform for now, to match fftw's REDFT10
      // 1 Autocorrelation from global powerspectrum at fft_num*ac_fft_len  (fft_num*ul_NumDataPoints )
      dim3 block(RPI, 1, 1);
      //  dim3 grid((ac_fftlen + (block.x*RPIB) - 1) / (block.x*RPIB), 1, 1); 
      dim3 grid(RPIB, (ac_fftlen + (block.x*RPIB) - 1) / (block.x*RPIB), 1); 
      
      //Step 1: Preprocessing - repack relevant powerspectrum into a 4N array with 'real-even symmetry'
      //////!!!  CUDA_ACC_SAFE_LAUNCH( (ac_RepackInputKernelR<<<grid, block>>>( &dev_PowerSpectrum[ac_fftlen*fft_num], (float2 *)dev_AutoCorrInR )),true);
      //  CUDA_ACC_SAFE_LAUNCH( (ac_RepackInputKernel<<<grid, block>>>( &dev_PowerSpectrum[ac_fftlen*fft_num], dev_AutoCorrIn )),true);
      CUDA_ACC_SAFE_LAUNCH( (ac_RepackInputKernelP<<<grid, block>>>( &dev_PowerSpectrum[ac_fftlen*fft_num], dev_AutoCorrIn[fft_num] )),true);
      
      //Step 2: Process the 4N-FFT (Complex to Complex, size is 4 * ac_fft_len)
      //////!!!  CUFFT_SAFE_CALL(hipfftExecR2C(cudaAutoCorr_planR, (float*)dev_AutoCorrInR , dev_AutoCorrOutR)); //, HIPFFT_FORWARD
      hipfftExecC2C(cudaAutoCorr_plan, dev_AutoCorrIn[fft_num] , dev_AutoCorrOut[fft_num], HIPFFT_FORWARD);
      
      
      //Step 3: Postprocess the FFT result (Scale, take powers & normalise), discarding unused data packing into AutoCorr_in first half for VRAM reuse
      //  block.x = RPS;
      //  block.y = 1;
      //  dim3 grid2( ((ac_fftlen>>1)+block.x*B-1)/(block.x*B), 1, 1);
      //  CUDA_ACC_SAFE_LAUNCH( (ac_RepackScaleKernel<<<grid2, block>>>( dev_AutoCorrOut, dev_AutoCorrIn )),true);
      block.x = RPS;
      block.y = 1;
      dim3 grid2(B, ((ac_fftlen>>1)+block.x*B-1)/(block.x*B), 1);
      CUDA_ACC_SAFE_LAUNCH( (ac_RepackScaleKernelP<<<grid2, block>>>( dev_AutoCorrOut[fft_num], dev_AutoCorrIn[fft_num])),true); //R first

      int len = ac_fftlen/2;
      int blksize = RDP; 
      dim3 block3(blksize,1,1);
      dim3 grid3(len/blksize,1,1);

      hipFuncSetCacheConfig(reinterpret_cast<const void*>(ac_reducePartial), hipFuncCachePreferShared);
      CUDA_ACC_SAFE_LAUNCH( (ac_reducePartial<<<grid3, block3, blksize*sizeof(float3)>>>( (float *)dev_AutoCorrIn[fft_num], dev_ac_partials[fft_num] )),true);
      blockSums[fft_num][0].x = -999;
      hipMemcpyAsync(&blockSums[fft_num][0], dev_ac_partials[fft_num], len/RDP*sizeof(float3), hipMemcpyDeviceToHost);

    }
 return 0;
}


// TODO (half done): start all autocorrs. start all datadownloads. do ALL peak finds. cudasync. process all autocorr results.

int cudaAcc_GetAutoCorrelation(float *AutoCorrelation, int ac_fftlen, int fft_num)
{
  int len = ac_fftlen/2;
  int blksize = RDP; 
  float rac_TotalSum = 0, ac_Peak = 0;
  int ac_PeakBin = 0;

  if(blockSums[fft_num][0].x == -999) { CUDASYNC; }
  for(int b = 0; b < len/blksize; b++)
    {
      rac_TotalSum += blockSums[fft_num][b].x;
      if(blockSums[fft_num][b].y > ac_Peak)
	{
	  ac_Peak = blockSums[fft_num][b].y;
	  ac_PeakBin = b;
	}
    }
  blockSums[fft_num][0].x = rac_TotalSum;
  blockSums[fft_num][0].y = ac_Peak;
  blockSums[fft_num][0].z = blockSums[fft_num][ac_PeakBin].z;
  
  return 0;
}

#endif //USE_CUDA
