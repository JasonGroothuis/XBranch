
#include <hip/hip_runtime.h>
#define B 2

__global__ void cudaAcc_GetPowerSpectrum_kernel( int NumDataPoints, float2* FreqData, float* PowerSpectrum) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;    
   
//	if (i < NumDataPoints) {
		float ax = FreqData[i].x;		
		float ay = FreqData[i].y;		
//		PowerSpectrum[i] = freqData.x * freqData.x + freqData.y * freqData.y;
		PowerSpectrum[i] = __fadd_rn( __fmul_rn(ax,ax),__fmul_rn(ay,ay)); 
//	}
}

/*
__global__ void cudaAcc_GetPowerSpectrum_kernel2( int NumDataPoints, float2* FreqData, float* PowerSpectrum) 
{
	const int i = blockIdx.x * blockDim.x*B + threadIdx.x;    

	float ax[B];
	float ay[B];
   
	#pragma unroll
	for (int k=0;k<B;k++)
	{
		ax[k] = FreqData[i+k*blockDim.x].x;		
		ay[k] = FreqData[i+k*blockDim.x].y;		
	}
//		PowerSpectrum[i] = freqData.x * freqData.x + freqData.y * freqData.y;

	#pragma unroll
	for (int k=0;k<B;k++)
	{
		PowerSpectrum[i+k*blockDim.x] = __fadd_rn( __fmul_rn(ax[k],ax[k]),__fmul_rn(ay[k],ay[k])); 
	}
}
*/

void cudaAcc_GetPowerSpectrum(int blksize, float2* dev_WorkData, float* dev_PowerSpectrum) {
    const int cudaAcc_NumDataPoints = 1024*1024;
	dim3 block(blksize, 1, 1);
	dim3 grid((cudaAcc_NumDataPoints + block.x - 1) / block.x, 1, 1);

	cudaAcc_GetPowerSpectrum_kernel<<<grid, block>>>(cudaAcc_NumDataPoints, dev_WorkData, dev_PowerSpectrum);
}

/*
void cudaAcc_GetPowerSpectrum2(int blksize, float2* dev_WorkData, float* dev_PowerSpectrum) {
    const int cudaAcc_NumDataPoints = 1024*1024;
	dim3 block(blksize, 1, 1);
	dim3 grid((cudaAcc_NumDataPoints + block.x*B - 1) / (block.x*B), 1, 1);

	cudaAcc_GetPowerSpectrum_kernel2<<<grid, block>>>(cudaAcc_NumDataPoints, dev_WorkData, dev_PowerSpectrum);
}
*/